#include "hip/hip_runtime.h"
/**
 * Timetravel-10 (bitcore) CUDA implementation
 *  by tpruvot@github - May 2017
 */

#include <stdio.h>
#include <memory.h>
#include <unistd.h>

#define HASH_FUNC_BASE_TIMESTAMP 1492973331U
#define HASH_FUNC_COUNT 10
#define HASH_FUNC_COUNT_PERMUTATIONS 40320U

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#if HASH_FUNC_COUNT > 10
#include "sph/sph_echo.h"
#endif
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

static uint32_t *d_hash[MAX_GPUS];
static uint32_t permutations[HASH_FUNC_COUNT_PERMUTATIONS] = {
	#include "timetravel-permutations.h"
};

enum Algo {
//	BLAKE = 0,
//	BMW,
	GROESTL = 0,
	SKEIN,
	JH,
	KECCAK,
	LUFFA,
	CUBEHASH,
	SHAVITE,
	SIMD,
	MAX_ALGOS_COUNT
};

#define INITIAL_DATE HASH_FUNC_BASE_TIMESTAMP
static inline uint32_t getCurrentAlgoSeq(uint32_t ntime)
{
	// unlike x11evo, the permutation changes often (with ntime)
	return (uint32_t) (ntime - INITIAL_DATE) % HASH_FUNC_COUNT_PERMUTATIONS;
}

// CPU Hash
extern "C" void bitcore_hash(void *output, const void *input)
{
	uint32_t _ALIGN(64) hash[64/4] = { 0 };

	sph_blake512_context     ctx_blake;
	sph_bmw512_context       ctx_bmw;
	sph_groestl512_context   ctx_groestl;
	sph_skein512_context     ctx_skein;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_luffa512_context     ctx_luffa1;
	sph_cubehash512_context  ctx_cubehash1;
	sph_shavite512_context   ctx_shavite1;
	sph_simd512_context      ctx_simd1;

	uint32_t *data = (uint32_t*)input;
	const uint32_t ntime = (opt_benchmark || !data[17]) ? (uint32_t)time(NULL) : data[17];
	const uint32_t sequence = permutations[getCurrentAlgoSeq(ntime)];

	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, hash, 64);
	sph_bmw512_close(&ctx_bmw, hash);

	for (int i = 0; i < (4 * (HASH_FUNC_COUNT - 2)); i += 4)
	{
		switch ((sequence >> i) & 0xf) {
		case GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, hash, 64);
			sph_groestl512_close(&ctx_groestl, hash);
			break;
		case SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, hash, 64);
			sph_skein512_close(&ctx_skein, hash);
			break;
		case JH:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, hash, 64);
			sph_jh512_close(&ctx_jh, hash);
			break;
		case KECCAK:
			sph_keccak512_init(&ctx_keccak);
			sph_keccak512(&ctx_keccak, hash, 64);
			sph_keccak512_close(&ctx_keccak, hash);
			break;
		case LUFFA:
			sph_luffa512_init(&ctx_luffa1);
			sph_luffa512(&ctx_luffa1, hash, 64);
			sph_luffa512_close(&ctx_luffa1, hash);
			break;
		case CUBEHASH:
			sph_cubehash512_init(&ctx_cubehash1);
			sph_cubehash512(&ctx_cubehash1, hash, 64);
			sph_cubehash512_close(&ctx_cubehash1, hash);
			break;
		case SHAVITE:
			sph_shavite512_init(&ctx_shavite1);
			sph_shavite512(&ctx_shavite1, hash, 64);
			sph_shavite512_close(&ctx_shavite1, hash);
			break;
		case SIMD:
			sph_simd512_init(&ctx_simd1);
			sph_simd512(&ctx_simd1, hash, 64);
			sph_simd512_close(&ctx_simd1, hash);
			break;
		}
	}

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "tt-"
#include "cuda_debug.cuh"

void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order);

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_bitcore(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 20 : 19;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark) pdata[17] = swab32(0x59090909);

	uint32_t ntime = swab32(work->data[17]);
	uint32_t sequence = permutations[getCurrentAlgoSeq(ntime)];

	if (opt_benchmark)
		ptarget[7] = 0x5;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffa512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		if (x11_simd512_cpu_init(thr_id, throughput) != 0) {
			return 0;
		}
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), -1);
		CUDA_CALL_OR_RET_X(hipMemset(d_hash[thr_id], 0, (size_t) 64 * throughput), -1);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	// first and second algo seems locked to blake+bmw in bitcore, fine!
	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		TRACE("blake  :");
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("bmw    :");

		for (int i = 0; i < (4 * (HASH_FUNC_COUNT - 2)); i += 4)
		{
			switch ((sequence >> i) & 0xf) {
			case GROESTL:
				quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("groestl:");
				break;
			case SKEIN:
				quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("skein  :");
				break;
			case JH:
				quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("jh512  :");
				break;
			case KECCAK:
				quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("keccak :");
				break;
			case LUFFA:
				x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("luffa  :");
				break;
			case CUBEHASH:
				x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("cube   :");
				break;
			case SHAVITE:
				x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("shavite:");
				break;
			case SIMD:
				x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("simd   :");
				break;
			}
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			const uint32_t Htarg = ptarget[7];
			be32enc(&endiandata[19], work->nonces[0]);
			bitcore_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				pdata[19] = work->nonces[0];
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					bitcore_hash(vhash, endiandata);
					if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
						bn_set_target_ratio(work, vhash, 1);
						work->valid_nonces++;
					}
					pdata[19] = max(pdata[19], work->nonces[1]) + 1;
				}
				return work->valid_nonces;
			} else if (vhash[7] > Htarg) {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_bitcore(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
